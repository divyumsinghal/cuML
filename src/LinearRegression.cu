#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <utils/cublas_utils.h>
#include <utils/cuda_utils.h>
#include <utils/cusolver_utils.h>

#include <include/LinearRegression.cuh>
#include <iostream>
#include <vector>

template <typename data_type>
LinearRegression<data_type>::LinearRegression() {
  CUBLAS_CHECK(hipblasCreate(&cublas_handle));
  CUSOLVER_CHECK(hipsolverDnCreate(&cusolver_handle));
}

template <typename data_type>
LinearRegression<data_type>::~LinearRegression() {
  if (cublas_handle) {
    hipblasDestroy(cublas_handle);
  }
  if (cusolver_handle) {
    hipsolverDnDestroy(cusolver_handle);
  }
}

template <typename data_type>
LinearRegression<data_type>::LinearRegression(
    LinearRegression &&other) noexcept {
  cublas_handle = other.cublas_handle;
  cusolver_handle = other.cusolver_handle;
  other.cublas_handle = nullptr;
  other.cusolver_handle = nullptr;
}

template <typename data_type>
LinearRegression<data_type> &LinearRegression<data_type>::operator=(
    LinearRegression &&other) noexcept {
  if (this != &other) {
    if (cublas_handle) {
      hipblasDestroy(cublas_handle);
    }
    if (cusolver_handle) {
      hipsolverDnDestroy(cusolver_handle);
    }
    cublas_handle = other.cublas_handle;
    cusolver_handle = other.cusolver_handle;
    other.cublas_handle = nullptr;
    other.cusolver_handle = nullptr;
  }
  return *this;
}

template <typename data_type>
void LinearRegression<data_type>::fit(std::vector<data_type> h_X,
                                      std::vector<data_type> h_y,
                                      std::vector<data_type> h_coefficients,
                                      int m, int n) {
  data_type *d_X, *d_y, *d_XTX, *d_XTy, *d_work;
  int *d_info;
  int lwork;

  // Allocate device memory
  hipMalloc(&d_X, m * n * sizeof(data_type));
  hipMalloc(&d_y, m * sizeof(data_type));
  hipMalloc(&d_XTX, n * n * sizeof(data_type));
  hipMalloc(&d_XTy, n * sizeof(data_type));
  hipMalloc(&d_info, sizeof(int));

  // Query workspace size for Cholesky decomposition
  hipsolverDnSpotrf_bufferSize(cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, d_XTX,
                              n, &lwork);

  hipMalloc(&d_work, lwork * sizeof(data_type));

  // Copy data to device
  hipMemcpy(d_X, h_X.data(), m * n * sizeof(data_type),
             hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y.data(), m * sizeof(data_type), hipMemcpyHostToDevice);

  const data_type alpha = 1.0f, beta = 0.0f;

  // 1. Compute X^T * X using hipblasSgemm
  hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, m, &alpha, d_X, m,
              d_X, m, &beta, d_XTX, n);

  // 2. Compute X^T * y using hipblasSgemv
  hipblasSgemv(cublas_handle, HIPBLAS_OP_T, m, n, &alpha, d_X, m, d_y, 1, &beta,
              d_XTy, 1);

  // 3. Solve (X^T * X) * coefficients = X^T * y using Cholesky decomposition
  hipsolverDnSpotrf(cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, d_XTX, n, d_work,
                   lwork, d_info);

  hipsolverDnSpotrs(cusolver_handle, HIPBLAS_FILL_MODE_LOWER, n, 1, d_XTX, n,
                   d_XTy, n, d_info);

  // Copy results back
  hipMemcpy(h_coefficients.data(), d_XTy, n * sizeof(data_type),
             hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_X);
  hipFree(d_y);
  hipFree(d_XTX);
  hipFree(d_XTy);
  hipFree(d_work);
  hipFree(d_info);
}

// Explicit template instantiation
template class LinearRegression<float>;